#include "hip/hip_runtime.h"
#include <stdio.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include <math.h>
#include <sys/time.h>
#include <iostream>
#include <assert.h>
#include <getopt.h>
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <string.h>
using namespace std;

#define PIXEL_WIDTH 3
#define TRESHOLD 4




#define SIZE    (100*1024*1024)
#define ELEMENTS    (SIZE / sizeof(unsigned int))
#define HASH_ENTRIES     2048

struct Entry {
    unsigned int    key;
    int            value;
    Entry           *next;
};

struct Table {
    size_t  count;
    Entry   **entries;
    Entry   *pool;
    Entry   *firstFree;
};

size_t hash( unsigned int key, size_t count ) {
    return key % count;
}

void initialize_table( Table &table, int entries, int elements ) {
    table.count = entries;
    table.entries = (Entry**)calloc( entries, sizeof(Entry*) );
    table.pool = (Entry*)malloc( elements * sizeof( Entry ) );
    table.firstFree = table.pool;
}

void free_table( Table &table ) {
    free( table.entries );
    free( table.pool );
}

void add_to_table( Table &table, unsigned int key, int value ) {
    size_t hashValue = hash( key, table.count );
    Entry *location = table.firstFree++;
    location->key = key;
    location->value = value;
    location->next = table.entries[hashValue];
    table.entries[hashValue] = location;
}

int getFromTable( Table &table, unsigned int key) {
    size_t hashValue = hash( key, table.count );

    Entry  *current = table.entries[hashValue];
    while (current != NULL) {
        if (hash( current->key, table.count ) != hashValue){
            current = current->next;
        } else {
            return current->value;
        }
    }
    return 0;
}

static const double kMicro = 1.0e-6;
double getTime()
{
	struct timeval TV;
	struct timezone TZ;

	const int RC = gettimeofday(&TV, &TZ);
	if(RC == -1) {
		printf("ERROR: Bad call to gettimeofday\n");
		return(-1);
	}

	return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}

void cmdLine(int argc, char *argv[], char* i, char* output, int& d, int& num_threads, int& seed);
void doHashTableStuff(Table red, Table green, Table blue, int x, int y, int *labels, unsigned char *data, int seed);

int main(int argc,char **argv)
{
	int display=0;
	int num_threads=1;
	int seed=time(NULL);
	char *i = (char *)malloc(100*sizeof(char));
	char *output = (char *)malloc(100*sizeof(char));
	i = strcpy(i, "input.png");
	output = strcpy(output, "output.png");


	cmdLine( argc, argv, i, output, display, num_threads,seed);
	int x,y,n;
	printf("Reading image %s...\n", i);
	unsigned char *data = stbi_load(i, &x, &y, &n, 0);
	if (!data) {
		fprintf(stderr, "Couldn't load image.\n%s image not found.",i);
		return (-1);
	}

	printf("Running segmentation...\n");
	int *labels = (int *)malloc(sizeof(int)*x*y);
	for(int i = 0; i < y; i++){
		for(int j = 0; j < x; j++){
			int idx = (i*x+j);
			int idx3 = idx*PIXEL_WIDTH;
			labels[idx] = 0;
			if((int)data[idx3] == 0) continue;
			labels[idx] = idx + 1;
		}
	}

	Table red;
	Table green;
	Table blue;

	initialize_table( red, HASH_ENTRIES, ELEMENTS );
	initialize_table( green, HASH_ENTRIES, ELEMENTS );
	initialize_table( blue, HASH_ENTRIES, ELEMENTS );

	int maxN = max(x,y);
	int phases = (int) ceil(log(maxN)/log(2)) + 1;
	double t0 = getTime();
	for(int pp = 0; pp <= phases; pp++){
		for(int i = 0; i < y; i++){
			for(int j = 0; j < x; j++){
				int idx = i*x + j;
				int idx3 = idx*PIXEL_WIDTH;
				if (labels[idx] == 0) continue;
				int ll = labels[idx];

				if(i != y-1 && abs((int)data[((i+1)*x + j)*PIXEL_WIDTH] - (int)data[idx3]) < TRESHOLD)
					labels[idx] = max(labels[idx], labels[(i+1)*x + j]);

				if(i != 0 && abs((int)data[((i-1)*x + j)*PIXEL_WIDTH] - (int)data[idx3]) < TRESHOLD)
					labels[idx] = max(labels[idx], labels[(i-1)*x + j]);

				if(i != y-1 && j != x-1 && abs((int)data[((i+1)*x + j + 1)*PIXEL_WIDTH] - (int)data[idx3]) < TRESHOLD)
					labels[idx] = max(labels[idx], labels[(i+1) * x + j + 1]);

				if(i != 0 && j != x-1 && abs((int)data[((i-1)*x + j + 1)*PIXEL_WIDTH] - (int)data[idx3]) < TRESHOLD)
					labels[idx] = max(labels[idx], labels[(i-1) * x + j + 1]);

				if(i != y-1 && j!= 0 && abs((int)data[((i+1)*x + j - 1)*PIXEL_WIDTH] - (int)data[idx3]) < TRESHOLD)
					labels[idx] = max(labels[idx], labels[(i+1) * x + j - 1]);

				if(i != 0 && j != 0 && abs((int)data[((i-1)*x + j - 1)*PIXEL_WIDTH] - (int)data[idx3]) < TRESHOLD)
					labels[idx] = max(labels[idx], labels[(i-1) * x + j - 1]);

				if (j != 0 && abs((int)data[(i*x + j - 1)*PIXEL_WIDTH] - (int)data[idx3]) < TRESHOLD)
					labels[idx] = max(labels[idx], labels[i*x + j - 1]);

				if (j != x-1 && abs((int)data[(i*x + j + 1)*PIXEL_WIDTH] - (int)data[idx3]) < TRESHOLD)
					labels[idx] = max(labels[idx], labels[i*x + j + 1]);

				if (ll < labels[idx]) {
					if (labels[ll - 1] < labels[idx])
						labels[ll - 1] = labels[idx];
				}
			}

		}

		for (int i = 0; i < y; i++) {
			for (int j = 0; j < x; j++) {
				int idx = i*x + j;
				if (labels[idx] != 0) {
					labels[idx] = max(labels[idx], labels[labels[idx] - 1]);
				}
			}
		}
		if(display){
			doHashTableStuff(red, green, blue, x, y, labels, data, seed);
			//Do the displaying here!
		}
	}
	
	double t1 = getTime();

	double t2, t3;
	if(!display){
		t2 = getTime();
		doHashTableStuff(red, green, blue, x, y, labels, data, seed);
		t3 = getTime();
	}
	
	printf("Labeling time: %f\n",(t1-t0));
	printf("Coloring time: %f\n",(t3-t2));
	printf("Total time: %f\n", (t1-t0)+(t3-t2));
	printf("Writing segmented image...\n");

	printf("returned %d\n",stbi_write_png(output, x, y, n, data, 0));

	free_table( red );
	free_table( blue );
	free_table( green );
	free(labels);
	stbi_image_free(data);
	return(0);
}

void doHashTableStuff(Table red, Table green, Table blue, int x, int y, int *labels, unsigned char *data, int seed){
	srand(seed);
	for (int i = 0; i < y; i++) {
		for (int j = 0; j < x; j++) {
			int label = labels[i*x+j];

			add_to_table( red, label, (int) random()*255 );
			add_to_table( green, label, (int) random()*255 );
			add_to_table( blue, label, (int) random()*255 );

		}
	}

	for (int i = 0; i < y; i++) {
		for (int j = 0; j < x; j++) {
			int label = labels[i*x+j];
			data[(i*x+j)*PIXEL_WIDTH+0] = (char)getFromTable(red,label);
			data[(i*x+j)*PIXEL_WIDTH+1] = (char)getFromTable(blue,label);
			data[(i*x+j)*PIXEL_WIDTH+2] = (char)getFromTable(green,label);

		}
	}
}


void cmdLine(int argc, char *argv[], char* i, char* output, int& d, int& num_threads, int &s){
/// Command line arguments
 // Default value of the domain sizes
 static struct option long_options[] = {
        {"i", required_argument, 0, 'i'},
        {"o", required_argument, 0, 'o'},
        {"display", required_argument, 0, 'd'},
        {"numthreads", required_argument, 0, 't'},
        {"seed", required_argument, 0, 's'},
 };
    // Process command line arguments
 int ac;
 for(ac=1;ac<argc;ac++) {
    int c;
    while ((c=getopt_long(argc,argv,"i:o:d:t:s:",long_options,NULL)) != -1){
        switch (c) {

	    // Name of input image
            case 'i':
            	strcpy(i, optarg);
                break;

	    // Nuber of threads
            case 't':
                num_threads = atoi(optarg);
                break;

	    // Turn on display
            case 'd':
                d = atoi(optarg);
                break;

	    // Output file name
            case 'o':
                output = strcpy(output, optarg);
                break;

        //Random seed
             case 's':
              	s = atoi(optarg);
               	break;

	    // Error
            default:
                printf("Usage: a.out [-i <input image name with format>] [-t <number of threads>]\n\t [-d turn on display]\n\t[-o <output file name with format>] [-s <random seed>]\n");
                exit(-1);
            }
    }
 }
}
