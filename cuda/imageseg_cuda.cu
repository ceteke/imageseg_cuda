#include "hip/hip_runtime.h"
#include <stdio.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include <math.h>
#include <sys/time.h>
#include <iostream>
#include <assert.h>
#include <getopt.h>
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <string.h>
#include <omp.h>
using namespace std;


#define PIXEL_WIDTH 3
#define TRESHOLD 4
#define BLOCK_SIZE 16

#define SIZE    (1024*1024*1024)
#define ELEMENTS    (SIZE / sizeof(unsigned int))
#define HASH_ENTRIES     1024

struct Entry {
	unsigned int    key;
	int            value;
	Entry           *next;
};

struct Table {
	size_t  count;
	Entry   **entries;
	Entry   *pool;
	Entry   *firstFree;
};

size_t hash( unsigned int key, size_t count ) {
	return key % count;
}

void initialize_table( Table &table, int entries, int elements ) {
	table.count = entries;
	table.entries = (Entry**)calloc( entries, sizeof(Entry*) );
	table.pool = (Entry*)malloc( elements * sizeof( Entry ) );
	table.firstFree = table.pool;
}

void free_table( Table &table ) {
	free( table.entries );
	free( table.pool );
}

void add_to_table( Table &table, unsigned int key, int value ) {
	size_t hashValue = hash( key, table.count );
	Entry *location = table.firstFree++;
	location->key = key;
	location->value = value;
	location->next = table.entries[hashValue];
	table.entries[hashValue] = location;
}

int getFromTable( Table &table, unsigned int key) {
	size_t hashValue = hash( key, table.count );

	Entry  *current = table.entries[hashValue];
	while (current != NULL) {
		if (hash( current->key, table.count ) != hashValue){
			current = current->next;
		} else {
			return current->value;
		}
	}
	return 0;
}

static const double kMicro = 1.0e-6;
double getTime()
{
	struct timeval TV;
	struct timezone TZ;

	const int RC = gettimeofday(&TV, &TZ);
	if(RC == -1) {
		printf("ERROR: Bad call to gettimeofday\n");
		return(-1);
	}

	return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}

void cmdLine(int argc, char *argv[], char* i, char* output, int& d, int& num_threads, int &s);
void doHashTableStuff(Table red, Table green, Table blue, int x, int y, int *labels, unsigned char *data, int seed, int num_threads);

__global__ void devicePhase(int *d_labels, unsigned char *d_data, int x, int y){
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= y ||  j >= x)return;

	int idx = i * x + j;
	int idx3 = idx*PIXEL_WIDTH;

	if (d_labels[idx] != 0){
		int ll = d_labels[idx];
		int dat = (int)d_data[idx3];

		if(i != y-1 && abs((int)d_data[((i+1)*x + j)*PIXEL_WIDTH] - dat) < TRESHOLD)
			d_labels[idx] = max(d_labels[idx], d_labels[(i+1)*x + j]);

		if(i != 0 && abs((int)d_data[((i-1)*x + j)*PIXEL_WIDTH] - dat) < TRESHOLD)
			d_labels[idx] = max(d_labels[idx], d_labels[(i-1)*x + j]);

		if(i != y-1 && j != x-1 && abs((int)d_data[((i+1)*x + j + 1)*PIXEL_WIDTH] - dat) < TRESHOLD)
			d_labels[idx] = max(d_labels[idx], d_labels[(i+1) * x + j + 1]);

		if(i != 0 && j != x-1 && abs((int)d_data[((i-1)*x + j + 1)*PIXEL_WIDTH] - dat) < TRESHOLD)
			d_labels[idx] = max(d_labels[idx], d_labels[(i-1) * x + j + 1]);

		if(i != y-1 && j!= 0 && abs((int)d_data[((i+1)*x + j - 1)*PIXEL_WIDTH] - dat) < TRESHOLD)
			d_labels[idx] = max(d_labels[idx], d_labels[(i+1) * x + j - 1]);

		if(i != 0 && j != 0 && abs((int)d_data[((i-1)*x + j - 1)*PIXEL_WIDTH] - dat) < TRESHOLD)
			d_labels[idx] = max(d_labels[idx], d_labels[(i-1) * x + j - 1]);

		if (j != 0 && abs((int)d_data[(i*x + j - 1)*PIXEL_WIDTH] - dat) < TRESHOLD)
			d_labels[idx] = max(d_labels[idx], d_labels[i*x + j - 1]);

		if (j != x-1 && abs((int)d_data[(i*x + j + 1)*PIXEL_WIDTH] - dat) < TRESHOLD)
			d_labels[idx] = max(d_labels[idx], d_labels[i*x + j + 1]);

		int label = d_labels[idx];

		if (ll < label) {
			if (d_labels[ll - 1] < label)
				d_labels[ll - 1] = label;
		}

		__syncthreads();

		if (label != 0) {
			d_labels[idx] = max(label, d_labels[label - 1]);
		}
	}

}

int main(int argc,char **argv)
{	
	int display=0;
	int num_threads=1;
	int seed=time(NULL);
	char *i = (char *)malloc(100*sizeof(char));
	char *output = (char *)malloc(100*sizeof(char));
	i = strcpy(i, "input.png");
	output = strcpy(output, "output.png");

	cmdLine( argc, argv, i, output, display, num_threads,seed);
	int x,y,n;
	printf("Reading image...%s\n", i);
	unsigned char *data = stbi_load(i, &x, &y, &n, 0);
	if (!data) {
		fprintf(stderr, "Couldn't load image %s.\n", i);
		return (-1);
	}
	int *labels = (int *)malloc(sizeof(int)*x*y);

	int *d_labels;
	unsigned char *d_data;

	hipMalloc((void**)&d_labels, sizeof(int)*x*y);
	hipMalloc((void**)&d_data, sizeof(unsigned char)*x*y*PIXEL_WIDTH);

	for(int i = 0; i < y; i++){
		for(int j = 0; j < x; j++){
			int idx = (i*x+j);
			int idx3 = idx*PIXEL_WIDTH;
			labels[idx] = 0;
			if((int)data[idx3] == 0) continue;
			labels[idx] = idx + 1;
		}
	}

	Table red;
	Table green;
	Table blue;

	initialize_table( red, HASH_ENTRIES, ELEMENTS );
	initialize_table( green, HASH_ENTRIES, ELEMENTS );
	initialize_table( blue, HASH_ENTRIES, ELEMENTS );

	int maxN = max(x,y);
	int phases = (int) ceil(log(maxN)/log(2)) + 1;

	hipMemcpy(d_data, data, sizeof(unsigned char)*x*y*PIXEL_WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(d_labels, labels, sizeof(int)*x*y, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 grid(ceil((double) x / BLOCK_SIZE), ceil((double) y / BLOCK_SIZE));

	printf("Applying segmentation...\n");
	hipEventRecord(start);
	for(int pp = 0; pp <= phases; pp++){
		devicePhase<<<grid, threads>>>(d_labels, d_data, x, y);
		if(display){
			hipMemcpy(labels, d_labels, sizeof(int)*x*y, hipMemcpyDeviceToHost);
			doHashTableStuff(red, green, blue, x, y, labels, data, seed, num_threads);
			//Burada display olacak.
		}
	}
	double t0, t1;
	hipEventRecord(stop);
	if(!display){
		hipMemcpy(labels, d_labels, sizeof(int)*x*y, hipMemcpyDeviceToHost);
		t0 = getTime();
		doHashTableStuff(red, green, blue, x, y, labels, data, seed, num_threads);
		t1 = getTime();
	}
	
	hipEventSynchronize(stop);
	hipFree(d_labels);
	hipFree(d_data);
	float kernelTime = 0;
	hipEventElapsedTime(&kernelTime, start, stop);

	printf("Kernel time: %f\n", kernelTime / 1000);
	printf("Coloring time: %f\n", t1-t0);
	printf("Total segmentation time: %f\n", (t1-t0) + (kernelTime / 1000));
	printf("Writing segmented image...\n");

	printf("returned %d\n",stbi_write_png(output, x, y, n, data, 0));

	free_table( red );
	free_table( blue );
	free_table( green );

	stbi_image_free(data);
	return(0);
}

void cmdLine(int argc, char *argv[], char* i, char* output, int& d, int& num_threads, int &s){
/// Command line arguments
 // Default value of the domain sizes
 static struct option long_options[] = {
        {"i", required_argument, 0, 'i'},
        {"o", required_argument, 0, 'o'},
        {"display", required_argument, 0, 'd'},
        {"numthreads", required_argument, 0, 't'},
        {"seed", required_argument, 0, 's'},
 };
    // Process command line arguments
 int ac;
 for(ac=1;ac<argc;ac++) {
    int c;
    while ((c=getopt_long(argc,argv,"i:o:d:t:s:",long_options,NULL)) != -1){
        switch (c) {

	    // Name of input image
            case 'i':
            	strcpy(i, optarg);
                break;

	    // Nuber of threads
            case 't':
                num_threads = atoi(optarg);
                break;

	    // Turn on display
            case 'd':
                d = atoi(optarg);
                break;

	    // Output file name
            case 'o':
                output = strcpy(output, optarg);
                break;

        //Random seed
             case 's':
              	s = atoi(optarg);
               	break;

	    // Error
            default:
                printf("Usage: a.out [-i <input image name with format>] [-t <number of threads>]\n\t [-d turn on display]\n\t[-o <output file name with format>] [-s <random seed>]\n");
                exit(-1);
            }
    }
 }
}

void doHashTableStuff(Table red, Table green, Table blue, int x, int y, int *labels, unsigned char *data, int seed, int num_threads){
	srand(seed);
	int i, j;

	for (i = 0; i < y; i++) {
		for (j = 0; j < x; j++) {
			int label = labels[i*x+j];
			add_to_table( red, label, (int)rand()*255);
			add_to_table( green, label, (int)rand()*255);
			add_to_table( blue, label, (int)rand()*255);
		}
	}
	
	#pragma omp parallel for num_threads(num_threads) private(i,j) schedule(dynamic)
	for (i = 0; i < y; i++) {
		for (j = 0; j < x; j++) {
			int idx = i*x+j;
			int idx3 = idx*PIXEL_WIDTH;
			int label = labels[idx];
			data[idx3+0] = (char)getFromTable(red,label);
			data[idx3+1] = (char)getFromTable(blue,label);
			data[idx3+2] = (char)getFromTable(green,label);

		}
	}
}

